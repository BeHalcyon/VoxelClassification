#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>


const int x26[26] = { -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1 };
const int y26[26] = { -1, -1, -1,  0,  0,  0,  1,  1,  1, -1, -1, -1,  0,  0,  1,  1,  1, -1, -1, -1,  0,  0,  0,  1,  1,  1 };
const int z26[26] = { -1, -1, -1, -1, -1, -1, -1, -1, -1,  0,  0,  0,  0,  0,  0,  0,  0,  1,  1,  1,  1,  1,  1,  1,  1,  1 };

const int x6[6] = { -1,  1,  0,  0,  0,  0 };
const int y6[6] = { 0,  0, -1,  1,  0,  0 };
const int z6[6] = { 0,  0,  0,  0, -1,  1 };


__constant__ int dx26[26];
__constant__ int dy26[26];
__constant__ int dz26[26];

__constant__ int dx6[6];
__constant__ int dy6[6];
__constant__ int dz6[6];


__global__
void kernel_segmentation(unsigned char* d_volume_data, const int sz, const int xDim, const int yDim, const int zDim, const float threshold, const int window_size,
	float* word_vec, size_t word_vecotr_size, int word_number, 
	float* label_vec,
                         size_t label_vector_size, size_t label_number, const int vector_size, int* segmentation_data)
{

	const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	//printf("%d\n", index);
	if (index >= sz) return;

	
	//std::vector<float> average_vector(vector_size, 0);

	float average_vector[100];
	for(auto i=0;i<100;i++)
	{
		average_vector[i] = 0;
	}


	const int oz = index / (xDim*yDim);
	const int ox = index % xDim;
	const int oy = (index % (xDim*yDim)) / xDim;
	int idx = 0;
	//std::stringstream ss;
	//ss << a;
	//std::string str = ss.str();
	auto center_word = d_volume_data[index];
	//const auto center_word_string = std::to_string(center_word);

	//float* word_vector = &word_vec[center_word*vector_size];

	for (auto j = 0; j < vector_size; j++) average_vector[j] += word_vec[center_word*vector_size+j];


	//const int dx26[26] = { -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1, -1,  0,  1 };
	//const int dy26[26] = { -1, -1, -1,  0,  0,  0,  1,  1,  1, -1, -1, -1,  0,  0,  1,  1,  1, -1, -1, -1,  0,  0,  0,  1,  1,  1 };
	//const int dz26[26] = { -1, -1, -1, -1, -1, -1, -1, -1, -1,  0,  0,  0,  0,  0,  0,  0,  0,  1,  1,  1,  1,  1,  1,  1,  1,  1 };

	auto cnt = 1;
	int len = 26;
	if (window_size == 3) len = 26;
	else if (window_size == 1) len = 1;
	for (auto i = 0; i < len; i++)
	{
		int nx = ox;//new x
		int ny = oy;//new y
		int nz = oz;//new z
		if(window_size==3)
		{
			nx += dx26[i];
			ny += dy26[i];
			nz += dz26[i];
		}
		else if(window_size==1)
		{
			nx += dx6[i];
			ny += dy6[i];
			nz += dz6[i];
		}

		if (nx >= 0 && nx < xDim && ny >= 0 && ny < yDim && nz >= 0 && nz < zDim)
		{
			int nind = nz * xDim*yDim + ny * xDim + nx;

			cnt++;
			
			//const auto center_word_string = std::to_string(volume_data[center_word]);
			//auto& context_vector = word_map[std::to_string(volume_data[nind])];

			idx = d_volume_data[nind] * vector_size;

			for (auto j = 0; j < vector_size; j++) average_vector[j] += word_vec[idx + j];
		}
	}

	for (auto j = 0; j < vector_size; j++) average_vector[j] /= cnt;

	//�жϸ�������label����Ĺ�ϵ
	float max_value = -0xffffff;

	cnt = 0;

	for (auto n = 0; n < label_number; n++) {
		//float* label_vector = &label_vec[n*vector_size];
		auto buf = 0.0f;
		for (auto m = 0; m < vector_size; m++)
		{
			buf += average_vector[m] * label_vec[n*vector_size+m];
		}
		if (buf > max_value)
		{
			max_value = buf;
			cnt = n + 1;
		}
	}
	if (max_value < threshold)
	{
		cnt = 0;
	}
	segmentation_data[index] = 10 * cnt;

}


extern "C" void kernelSegmentation(int block_number, int thread_number, unsigned char* d_volume_data, const int sz, const int xDim, const int yDim, const int zDim,
	const float threshold, const int window_size,
	float* word_vector, size_t word_vecotr_size,
	int word_number, float* label_vector, size_t label_vector_size, size_t label_number, const int vector_size, int* segmentation_data)
{
	
	(hipMemcpyToSymbol(HIP_SYMBOL(dx26), x26, sizeof(int) * 26, 0, hipMemcpyHostToDevice));
	(hipMemcpyToSymbol(HIP_SYMBOL(dy26), y26, sizeof(int) * 26, 0, hipMemcpyHostToDevice));
	(hipMemcpyToSymbol(HIP_SYMBOL(dz26), z26, sizeof(int) * 26, 0, hipMemcpyHostToDevice));

	(hipMemcpyToSymbol(HIP_SYMBOL(dx6), x6, sizeof(int) * 6, 0, hipMemcpyHostToDevice));
	(hipMemcpyToSymbol(HIP_SYMBOL(dy6), y6, sizeof(int) * 6, 0, hipMemcpyHostToDevice));
	(hipMemcpyToSymbol(HIP_SYMBOL(dz6), z6, sizeof(int) * 6, 0, hipMemcpyHostToDevice));


	kernel_segmentation << <block_number, thread_number >> > (d_volume_data, sz, xDim, yDim, zDim, threshold,window_size, word_vector, word_vecotr_size,
		word_number, label_vector, label_vector_size, label_number, vector_size, segmentation_data);



	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		//goto Error;
	}
	hipDeviceSynchronize();
	//printf("Kernel end.\n");
}
